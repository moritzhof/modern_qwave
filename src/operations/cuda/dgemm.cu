#include "hip/hip_runtime.h"
#ifdef __HIPCC__

#include "cutlass/gemm/device/gemm.h"
#include <hipblas.h>
/*########################################################
      These implementations assuming column-major order
 ########################################################*/

namespace qwv{
 namespace cuda{
  
 __global__
 void dgemm1(int M, int N, int K, double alpha, double const *A, double const *B, double beta, double *C) {
     
     typedef block_task_policy < 128, 32, 8, 4,8, true, block_raster_enum::Default> block_task_policy_t;
     
     typedef gemm::blas_scaled_epilogue<double, double, double> gemm_op_t;
     
     typedef block_task< block_task_policy_t, double, double, matrix_transform_t::Transpose, 4, matrix_transform_t::NonTranspose, 4, gemm_op_t, 4, true > block_task_t;

     __shared__ block_task_t::scratch_storage_t smem;

     block_task_t(reinterpret_cast(&smem), &smem, A, B, C, gemm_op_t(alpha, beta), M, N, K).run();
}
 
 
 __global__
 void dgemm2(int M, int N, int K, double alpha, double const *A, int lda,
  double const *B, int ldb, double beta, double *C, int ldc) {

     typedef block_task_policy < 128, 32, 8, 4,8, true, block_raster_enum::Default> block_task_policy_t;
     
     typedef gemm::blas_scaled_epilogue<double, double, double> gemm_op_t;
     
     typedef block_task< block_task_policy_t,double,double,matrix_transform_t::NonTranspose,4, matrix_transform_t::NonTranspose, 4, gemm_op_t, 4, true > block_task_t ;

     __shared__ block_task_t::scratch_storage_t smem;

     block_task_t(reinterpret_cast(&smem), &smem, A, B, C, gemm_op_t(alpha, beta), M, N, K).run();

 } // end of cuda namespace
} // end of qwv namespace

namespace qwv{
 namespace cublas{
 void dgemm1(const qwv::cuda::device_ptr<double> A, const qwv::cuda::device_ptr<double> B, qwv::cuda::device_ptr<T> C,
            const int m, const int k, const int n) {
       int lda=m,ldb=k,ldc=m;
       const double alf = 1;
       const double bet = 0;
       const double *alpha = &alf;
       const double *beta = &bet;
 

      hipblasHandle_t handle;
      hipblasCreate(&handle);
 
      hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
 
     hipblasDestroy(handle);
    }
 
 
 void dgemm2(const qwv::cuda::device_ptr<double> A, const qwv::cuda::device_ptr<double> B, qwv::cuda::device_ptr<T> C,
            const int m, const int k, const int n) {
     
       int lda=m,ldb=k,ldc=m;
       const double alf = 1;
       const double bet = 0;
       const double *alpha = &alf;
       const double *beta = &bet;
 

      hipblasHandle_t handle;
      hipblasCreate(&handle);
 
      hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha, A, lda, B, ldb, beta, C, ldc);
 
     hipblasDestroy(handle);
    }
 } // end of cuda namespace
} // end of qwv namespace
#endif
